#include "hip/hip_runtime.h"
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__

// This is slightly mental, but gets it to properly index device function calls like __popc and whatever.
#define __HIPCC__

#include <hip/device_functions.h>

// These headers are all implicitly present when you compile CUDA with clang. Clion doesn't know that, so
// we include them explicitly to make the indexer happy. Doing this when you actually build is, obviously,
// a terrible idea :D
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>

#endif // __JETBRAINS_IDE__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define BLOCKDIM_X 32
#define BLOCKDIM_Y 32
#define BLOCKDIM BLOCKDIM_X * BLOCKDIM_Y
#define W (512 * 200)
#define H 512
#define N W * H

#define CUDA_CALL(F, ...)\
    if((F(__VA_ARGS__)) != hipSuccess){\
        hipError_t e = hipGetLastError();\
        printf("CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
        return(EXIT_FAILURE);\
    }

#define CURAND_CALL(F, ...)\
    if((F(__VA_ARGS__)) != HIPRAND_STATUS_SUCCESS){\
        hipError_t e = hipGetLastError();\
        if(e != hipSuccess){\
            printf("CuRAND failure %s:%d: '%s'\n",__FILE__,__LINE__, hipGetErrorString(e));\
        }\
        return(EXIT_FAILURE);\
    }

#define PRINT_1D(A, S)\
    printf("[");\
    for(int i = 0; i < S; i++){\
        printf("%f, ", A[i]);\
    }\
    printf("]\n");

#define PRINT_FLAT2D(A, WIDTH, HEIGHT)\
    printf("[\n");\
    for(int i = 0; i < WIDTH; i++){\
        printf("[");\
        for(int j = 0; j < HEIGHT; j++){\
            printf("%f, ", A[i + j * WIDTH]);\
        }\
        printf("]\n");\
    }\
    printf("]\n");

#define KERNEL_R 3

__global__ void fill(float *a, float v) {
    size_t x = threadIdx.x;
    size_t y = blockIdx.x;
    if (x < W && y < H)
        a[x + y * W] = v;
}

__global__ void sobelOperator(const float *src, int w, int h, const float th, float *dst){
    float gx[9] = {
            1, 0, -1,
            2, 0, -2,
            1, 0, -1
    };
    float gy[9] = {
            1, 2, 1,
            0, 0, 0,
            -1, -2, -1
    };
    int i, j, k, l;
    int r = 3;
    int rHalf = r / 2;

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x < w && y < h) {
        float gxMag = 0.0f;
        float gyMag = 0.0f;
        for (i = x - rHalf, k = 0; i <= x + rHalf; i++, k++) {
            for (j = y - rHalf, l = 0; j <= y + rHalf; j++, l++) {
                float value = 0.0f;
                if (0 <= i && i < w && 0 <= j && j < h) {
                    value = src[i + j * w];
                }
                gxMag += value * gx[k + l * r];
                gyMag += value * gy[k + l * r];
            }
        }
        float mag = __fsqrt_rd(gxMag * gxMag + gyMag * gyMag);
        dst[x + y * w] = mag >= th ? 1.0f : 0.0f;
    }
}

int main() {
    //Host variables
    float *img;
    float *res;

    //Device variables
    float *deviceImg;
    float *deviceRes;

    hiprandGenerator_t gen;

    img = (float *) malloc(sizeof(float) * N);
    res = (float *) malloc(sizeof(float) * N);

    CUDA_CALL(hipMalloc, (void **) &deviceImg, sizeof(float) * N);
    CUDA_CALL(hipMalloc, (void **) &deviceRes, sizeof(float) * N);

    CURAND_CALL(hiprandCreateGenerator, &gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed, gen, 1234ULL);

    dim3 blockSize = dim3(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 gridSize = dim3((unsigned int) ceil(W / BLOCKDIM_X), (unsigned int) ceil(H / BLOCKDIM_Y));
    CURAND_CALL(hiprandGenerateUniform, gen, deviceImg, N);
//    fill << < (BLOCKDIM + N) / BLOCKDIM, BLOCKDIM >> > (deviceImg, 1.0);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sobelOperator <<<gridSize, blockSize >>> (deviceImg, W, H, 0.5, deviceRes);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    CUDA_CALL(hipMemcpy, img, deviceImg, N * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CALL(hipMemcpy, res, deviceRes, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Source image: \n");
    PRINT_FLAT2D(img, W, H);

    printf("Blurred image: \n");
    PRINT_FLAT2D(res, W, H);

    printf("TIME = %f\n", milliseconds / 1000.0f);

    CURAND_CALL(hiprandDestroyGenerator, gen);
    CUDA_CALL(hipFree, deviceImg);
    CUDA_CALL(hipFree, deviceRes);

    CUDA_CALL(hipEventDestroy, start);
    CUDA_CALL(hipEventDestroy, stop);

    CUDA_CALL(hipDeviceReset);
    free(img);
    free(res);

    return EXIT_SUCCESS;
}