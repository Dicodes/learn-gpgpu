#include "hip/hip_runtime.h"
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__

// This is slightly mental, but gets it to properly index device function calls like __popc and whatever.
#define __HIPCC__

#include <hip/device_functions.h>

// These headers are all implicitly present when you compile CUDA with clang. Clion doesn't know that, so
// we include them explicitly to make the indexer happy. Doing this when you actually build is, obviously,
// a terrible idea :D
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>

#endif // __JETBRAINS_IDE__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define BLOCKDIM_X 32
#define BLOCKDIM_Y 32
#define BLOCKDIM BLOCKDIM_X * BLOCKDIM_Y
#define W (512 * 200)
#define H 512
#define N W * H

#define CUDA_CALL(F, ...)\
    if((F(__VA_ARGS__)) != hipSuccess){\
        hipError_t e = hipGetLastError();\
        printf("CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
        return(EXIT_FAILURE);\
    }

#define CURAND_CALL(F, ...)\
    if((F(__VA_ARGS__)) != HIPRAND_STATUS_SUCCESS){\
        hipError_t e = hipGetLastError();\
        if(e != hipSuccess){\
            printf("CuRAND failure %s:%d: '%s'\n",__FILE__,__LINE__, hipGetErrorString(e));\
        }\
        return(EXIT_FAILURE);\
    }

#define PRINT_1D(A, S)\
    printf("[");\
    for(int i = 0; i < S; i++){\
        printf("%f, ", A[i]);\
    }\
    printf("]\n");

#define PRINT_FLAT2D(A, WIDTH, HEIGHT)\
    printf("[\n");\
    for(int i = 0; i < WIDTH; i++){\
        printf("[");\
        for(int j = 0; j < HEIGHT; j++){\
            printf("%f, ", A[i + j * WIDTH]);\
        }\
        printf("]\n");\
    }\
    printf("]\n");

#define KERNEL_R 3

__global__ void fill(float *a, float v) {
    size_t x = threadIdx.x;
    size_t y = blockIdx.x;
    if (x < W && y < H)
        a[x + y * W] = v;
}

__device__ void sort(float *a, float *b, float *c) {
    float d;
    if(*a > *b){
        d = *a;
        *a = *b;
        *b  = d;
    }
    if(*a > *c){
        d = *a;
        *a = *c;
        *c  = d;
    }
    if(*b > *c){
        d = *b;
        *b = *c;
        *c  = d;
    }
}

__device__ void swap(float *a, float *b){
    float d = *a;
    *a = *b;
    *b = d;
}

__device__ void insertionSort(float *src, int n){
    for(int i = 1; i < n; i++){
        int j = i - 1;
        float key = src[i];
        while(j >= 0 && src[j] > key){
            if(src[j] > src[j + 1])
                swap(&src[j], &src[j + 1]);
            j--;
        }
    }
}

__global__ void naiveMedianFilter(const float *src, int w, int h, float *dst){
    float imgBlock float[KERNEL_R * KERNEL_R];
    int r = KERNEL_R;
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int rHalf = r / 2;
    int i, j, k, l;
    if(x - rHalf > 0 && x + rHalf < w && y - rHalf > 0 && y + rHalf < h){
        for (i = x - rHalf, k = 0; i <= x + rHalf; i++, k++) {
            for (j = y - rHalf, l = 0; j <= y + rHalf; j++, l++) {
                imgBlock[k + l * r] = src[i + j * w];
            }
        }

        //Sort block
        insertionSort(imgBlock, r * r);

        //Set median
        dst[x + y * w] = imgBlock[rHalf + rHalf * r];
    }
    else if(x < w && y < h){
        dst[x + y * w] = src[x + y * w];
    }
}


int main() {
    //Host variables
    float *img;
    float *res;

    //Device variables
    float *deviceImg;
    float *deviceRes;

    hiprandGenerator_t gen;

    img = (float *) malloc(sizeof(float) * N);
    res = (float *) malloc(sizeof(float) * N);

    CUDA_CALL(hipMalloc, (void **) &deviceImg, sizeof(float) * N);
    CUDA_CALL(hipMalloc, (void **) &deviceRes, sizeof(float) * N);

    CURAND_CALL(hiprandCreateGenerator, &gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed, gen, 1234ULL);

    dim3 blockSize = dim3(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 gridSize = dim3((unsigned int) ceil(W / BLOCKDIM_X), (unsigned int) ceil(H / BLOCKDIM_Y));
    CURAND_CALL(hiprandGenerateUniform, gen, deviceImg, N);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    medianFilter<<<gridSize, blockSize>>>(deviceImg, W, H, deviceRes);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds;
    hipEventElapsedTime(&milliseconds, start, stop);

    CUDA_CALL(hipMemcpy, img, deviceImg, N * sizeof(float), hipMemcpyDeviceToHost);
    CUDA_CALL(hipMemcpy, res, deviceRes, N * sizeof(float), hipMemcpyDeviceToHost);

    printf("Source image: \n");
    PRINT_FLAT2D(img, W, H);

    printf("Blurred image: \n");
    PRINT_FLAT2D(res, W, H);

    printf("TIME = %f\n", milliseconds / 1000.0f);

    CURAND_CALL(hiprandDestroyGenerator, gen);
    CUDA_CALL(hipFree, deviceImg);
    CUDA_CALL(hipFree, deviceRes);

    CUDA_CALL(hipEventDestroy, start);
    CUDA_CALL(hipEventDestroy, stop);

    CUDA_CALL(hipDeviceReset);
    free(img);
    free(res);

    return EXIT_SUCCESS;
}