// -*- mode: C -*-
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1024

#define FILL(A, V, S) \
	for(size_t i = 0; i < S; i++){\
		A[i] = V;\
	}\

#define SUM(ACC, A, S) \
	for(size_t i = 0; i < S; i++){\
		ACC += A[i]; \
	}\

__global__ void vecAdd(float* a, float *b, float *res, int n) { 
	int id = threadIdx.x + blockIdx.x * blockDim.x;
  if(id < n){
    res[id] = a[id] + b[id];
  }
}

int main(int argc, const char** argv) {

  float *a, *b, *c;
  size_t size = sizeof(float) * N;
  a = (float*) malloc(size);
  b = (float*) malloc(size);
  c = (float*) malloc(size);
  FILL(a, 1.0, N);
  FILL(b, 2.0, N);

  float *deviceA, *deviceB, *deviceC;
  hipMalloc((void**) &deviceA, size);
  hipMalloc((void**) &deviceB, size);
  hipMalloc((void**) &deviceC, size);

  hipMemcpy(deviceA, a, size, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, b, size, hipMemcpyHostToDevice);

  size_t gridSize = N >> 1;
  size_t blockSize = N >> 1;
  vecAdd <<<gridSize, blockSize>>> (deviceA, deviceB, deviceC, N);

  hipMemcpy(c, deviceC, size, hipMemcpyDeviceToHost);
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  float sum = 0.0;
  SUM(sum, c, N);

  printf("Sum = %f\n", sum);
  printf("Assert = %f\n", sum / 3.0 / N);
  free(a);
  free(b);
  free(c);
  return 0;  
}