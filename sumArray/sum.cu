#include "hip/hip_runtime.h"
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__

// This is slightly mental, but gets it to properly index device function calls like __popc and whatever.
#define __HIPCC__

#include <hip/device_functions.h>

// These headers are all implicitly present when you compile CUDA with clang. Clion doesn't know that, so
// we include them explicitly to make the indexer happy. Doing this when you actually build is, obviously,
// a terrible idea :D
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>

#endif // __JETBRAINS_IDE__

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdio.h>

#define CUDA_CALL(F, ...)\
    if((F(__VA_ARGS__)) != hipSuccess){\
        hipError_t e = hipGetLastError();\
        printf("Failure in CUDA %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));\
        return(EXIT_FAILURE);\
    }

#define CURAND_CALL(F, ...)\
    if((F(__VA_ARGS__)) != HIPRAND_STATUS_SUCCESS){\
        hipError_t e = hipGetLastError();\
        if(e != hipSuccess){\
            printf("CuRAND failure %s:%d: '%s'\n",__FILE__,__LINE__, hipGetErrorString(e));\
        }\
        return(EXIT_FAILURE);\
    }

#define PRINT_1D(A, S)\
    printf("[");\
    for(int i = 0; i < S; i++){\
        printf("%f, ", A[i]);\
    }\
    printf("]\n");

#define PRINT_FLAT2D(A, WIDTH, HEIGHT)\
    printf("[\n");\
    for(int i = 0; i < WIDTH; i++){\
        printf("[");\
        for(int j = 0; j < HEIGHT; j++){\
            printf("%f, ", A[i + j * WIDTH]);\
        }\
        printf("]\n");\
    }\
    printf("]\n");

#define W 20
#define H 20
#define N (W * H)
#define BLOCKDIM 1024
#define BLOCKDIM_X 32
#define BLOCKDIM_Y 32

__global__ void seq(float* dst, int w, int h){
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	dst[tid] = tid;
}

__global__ void memsetVarF32(float *src, float c){
	*src = c;
}

__global__ void sum(const float* src, float *s, int n){
	__shared__ float _s[BLOCKDIM]; 
	int tid = threadIdx.x;
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	if(id < n){
		_s[tid] = src[id];
	}
	else{
		_s[tid] = 0.0;
	}
	__syncthreads();
	for(unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2){
		if(tid < stride){
			_s[tid] += _s[tid + stride];
		}
	}
	__syncthreads();
	if(tid == 0){
		atomicAdd(s, _s[0]);
	}
}

int main(){
	hipEvent_t start, stop;
	CUDA_CALL(hipEventCreate, &start);
	CUDA_CALL(hipEventCreate, &stop);
	float *src;
	float *devSrc;
	float *devAvg;
	float *devS;
	float _avg = 0.0;
	src = (float*) malloc(N * sizeof(float));
	CUDA_CALL(hipMalloc, (void**) &devSrc, N * sizeof(float));
	CUDA_CALL(hipMalloc, (void**) &devAvg, sizeof(float));
	CUDA_CALL(hipMalloc, (void**) &devS, sizeof(float));

	dim3 blockSize(BLOCKDIM_X, BLOCKDIM_Y);
	dim3 gridSize((W + BLOCKDIM_X) / W, (H + BLOCKDIM_Y) / H);
	seq<<<(N + BLOCKDIM) / BLOCKDIM, BLOCKDIM>>>(devSrc, W, H);
	// memsetVarF32<<<1, 1>>>(devS, 0.0);

	hipEventRecord(start);
	memsetVarF32<<<1, 1>>>(devS, 0.0);
	sum<<<(N + BLOCKDIM) / BLOCKDIM, BLOCKDIM>>>(devSrc, devS, N);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	CUDA_CALL(hipMemcpy, src, devSrc, N * sizeof(float), hipMemcpyDeviceToHost);
	CUDA_CALL(hipMemcpy, &_avg, devS, sizeof(float), hipMemcpyDeviceToHost);
	// PRINT_1D(src, N);
	printf("AVG = %f\n", _avg);
	printf("TIME = %f (msec)\n", milliseconds);
	CUDA_CALL(hipFree, devSrc);
	CUDA_CALL(hipFree, devAvg);
	CUDA_CALL(hipFree, devS);
	CUDA_CALL(hipEventDestroy, start);
	CUDA_CALL(hipEventDestroy, stop);
	free(src);
	return EXIT_SUCCESS;
}